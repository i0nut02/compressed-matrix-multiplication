#include "../../include/cuda/cuda_mem.cuh"
#include "../../include/cuda/cuda_check.cuh"

void vector_malloc_cuda(float* d_vec, int numElements) {
    CHECK_CUDA_ERROR(hipMalloc((void**)d_vec, numElements * sizeof(float)));
}

void vector_copy_cuda(float* h_vec, float* d_vec, int numElements) {
    CHECK_CUDA_ERROR(hipMemcpy(d_vec, h_vec, numElements * sizeof(float), hipMemcpyHostToDevice));
}

void vector_copy_2host(float* h_vec, float* d_vec, int numElements) {
    CHECK_CUDA_ERROR(hipMemcpy(h_vec, d_vec, numElements, hipMemcpyDeviceToHost));
}

void vector_copy_cuda(int* h_vec, int* d_vec, int numElements) {
    CHECK_CUDA_ERROR(hipMemcpy(d_vec, h_vec, numElements * sizeof(int), hipMemcpyHostToDevice));
}

void vector_free_cuda(float* d_vector){
    CHECK_CUDA_ERROR(hipFree(d_vector));
}

void vector_free_cuda(int* d_vector){
    CHECK_CUDA_ERROR(hipFree(d_vector));
}
