#include "hip/hip_runtime.h"
#include "../../include/cuda/hyb_cuda.cuh"
#include "../../include/cuda/cuda_check.cuh"
#include "../../include/cuda/ell_cuda.cuh"

void allocate_hyp_memory_cuda(float** d_ell_values, int** d_ell_col_indices, int ell_elements, //
                              float** d_coo_values, int** d_coo_row_indices, int** d_coo_col_indices, int coo_elements) {
    CHECK_CUDA_ERROR(hipMalloc(d_ell_values, ell_elements * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(d_ell_col_indices, ell_elements * sizeof(int)));

    CHECK_CUDA_ERROR(hipMalloc(d_coo_values, coo_elements * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(d_coo_row_indices, coo_elements * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(d_coo_col_indices, coo_elements * sizeof(int)));
}

__global__ void _hyb_matrix_multiply_kernel_impl(const float* A_ellValues, const int* A_ellColIndices,
                                                 const float* B_ellValues, const int* B_ellColIndices,
                                                 float* C, int numRowsC, int numColsC,
                                                 int maxNumNonZeroA, int maxNumNonZeroB,
                                                 const float* A_cooValues, const int* A_cooRowIndices, const int* A_cooColIndices,
                                                 const float* B_cooValues, const int* B_cooRowIndices, const int* B_cooColIndices,
                                                 int A_cooElements, int B_cooElements)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < numRowsC * numColsC) {
        int row = id / numColsC;
        int col = id % numColsC;

        float sum = 0.0f;

        for (int i = 0; i < maxNumNonZeroA; i++) {
            int idxA = row * maxNumNonZeroA + i;
            if (A_ellValues[idxA] == 0) {
                continue;
            }
            
            int a_col_idx = A_ellColIndices[idxA];

            for (int j = 0; j < maxNumNonZeroB; j++) {
                int idxB = col * maxNumNonZeroB + j;
                
                if (B_ellValues[idxB] == 0) {
                    break;
                }
                
                int b_col_idx = B_ellColIndices[idxB];

                if (b_col_idx > a_col_idx) {
                    break;
                }
                
                if (b_col_idx == a_col_idx) {
                    sum += B_ellValues[idxB] * A_ellValues[idxA];
                }
            }
        }

        for (int i = 0; i < A_cooElements; i++) {
            if (A_cooRowIndices[i] == row) {
                for (int j = 0; j < B_cooElements; j++) {
                    if (B_cooRowIndices[j] == row && B_cooColIndices[j] == A_cooColIndices[i]) {
                        sum += A_cooValues[i] * B_cooValues[j];
                    }
                }
            }
        }
        
        C[row * numColsC + col] = sum;
    }
}

void hyb_matrix_multiply_launch(const float* A_ellValues, const int* A_ellColIndices,
                                const float* B_ellValues, const int* B_ellColIndices,
                                float* C, int numRowsC, int numColsC,
                                int maxNumNonZeroA, int maxNumNonZeroB,
                                const float* A_cooValues, const int* A_cooRowIndices, const int* A_cooColIndices,
                                const float* B_cooValues, const int* B_cooRowIndices, const int* B_cooColIndices,
                                int A_cooElements, int B_cooElements)
{
    int threadsPerBlock = 256;
    
    int totalElementsC = numRowsC * numColsC;
    
    int numBlocks = (totalElementsC + threadsPerBlock - 1) / threadsPerBlock;
    
    _hyb_matrix_multiply_kernel_impl<<<numBlocks, threadsPerBlock>>>(
        A_ellValues, A_ellColIndices,
        B_ellValues, B_ellColIndices,
        C, numRowsC, numColsC,
        maxNumNonZeroA, maxNumNonZeroB,
        A_cooValues, A_cooRowIndices, A_cooColIndices,
        B_cooValues, B_cooRowIndices, B_cooColIndices,
        A_cooElements, B_cooElements
    );
    
    CHECK_CUDA_ERROR(hipGetLastError());
    
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}